#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <vector>
#include <hipcub/hipcub.hpp>

constexpr int num_blocks = 2;
constexpr int warps_per_block = 4;
constexpr int threads_per_warp = 10;

using WarpScanner = hipcub::WarpScan<int, threads_per_warp>;

__global__ void warpScan(int* vec, int* out, int* agg) {
    // Allocate shared memory for thread communication                         
    __shared__ WarpScanner::TempStorage temp[warps_per_block];

    // Assign thread local variables and data
    if (threadIdx.x % 32 < threads_per_warp) {
        int warp_lid = threadIdx.x / 32;
        int warp_gid = blockIdx.x * warps_per_block + warp_lid;
        int thread_gid = warp_gid * threads_per_warp 
                       + threadIdx.x % 32;
        int thread_data = vec[thread_gid];
        int thread_prod, warp_aggr;
            
        // Compute scan inside each warp
        WarpScanner(temp[warp_lid]).InclusiveScan(
            thread_data, thread_prod, hipcub::Sum(), warp_aggr
        );

        // Write to output
        out[thread_gid] = thread_prod;
        agg[thread_gid] = warp_aggr;
    }
}

int main() {
    // Useful values
    constexpr int physical_threads_per_block = warps_per_block * 32;
    constexpr int logical_threads_per_block = warps_per_block * threads_per_warp;
    constexpr int allocation_size = num_blocks * logical_threads_per_block;
    
    // Create vectors on host
    std::vector<int> h_vec(allocation_size);
    std::vector<int> h_out(allocation_size);
    std::vector<int> h_agg(allocation_size);

    // Fill host input vector
    for (size_t i = 0; i < allocation_size; ++i) {
        h_vec[i] = i % threads_per_warp + 1.f;
    }

    // Allocate memory on device
    void *p_vec, *p_out, *p_agg;
    hipMalloc(&p_vec, allocation_size * sizeof(int));
    hipMalloc(&p_out, allocation_size * sizeof(int));
    hipMalloc(&p_agg, allocation_size * sizeof(int));
    int* d_vec = static_cast<int*>(p_vec);
    int* d_out = static_cast<int*>(p_out);
    int* d_agg = static_cast<int*>(p_agg);

    // Copy memory from host to device
    hipMemcpy(d_vec, h_vec.data(), allocation_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with num_warps warps
    warpScan<<<num_blocks, physical_threads_per_block>>>(d_vec, d_out, d_agg);

    // Check that execution went well, or print error string
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }

    // Copy result from device to host
    hipMemcpy(h_out.data(), d_out, allocation_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_agg.data(), d_agg, allocation_size * sizeof(int), hipMemcpyDeviceToHost);

    // Check results
    for (int i = 0; i < allocation_size; ++i) {
        const int j = i % threads_per_warp;
        if (j == 0) {
            std::cout << std::endl;
            std::cout << " Warp: " << i / threads_per_warp << "\n";
            std::cout << " INPUT     SUM    WARP\n";
            std::cout << "----------------------\n";
        }
        std::cout << std::setw(6) << h_vec[i] << "  "
                  << std::setw(6) << h_out[i] << "  " 
                  << std::setw(6) << h_agg[i] << std::endl;
    }

    // Free device memory and return
    hipFree(d_vec);
    hipFree(d_out);
    hipFree(d_agg);
    return 0;
}
