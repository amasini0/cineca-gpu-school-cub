#include <iomanip>
#include <iostream>
#include <numeric>
#include <vector>
#include <hipcub/hipcub.hpp>

struct SquareAndAdd {
    int val;
    HIPCUB_RUNTIME_FUNCTION explicit SquareAndAdd(int x) : val{x} {}
    __device__ void operator() (int& x) { x *= x; x += val; };
};

int main() {
    // Useful values
    constexpr int size = 10000;

    // Initialize host vector
    std::vector<int> numbers(size);
    std::vector<int> results(size);
    std::iota(numbers.begin(), numbers.end(), 0);
    
    // Allocate device memory and copy from host
    void* p_numbers;
    hipMalloc(&p_numbers, size * sizeof(int));
    int* d_numbers = static_cast<int*>(p_numbers);
    hipMemcpy(d_numbers, numbers.data(), size * sizeof(int), hipMemcpyHostToDevice);

    // DeviceFor application
    hipcub::DeviceFor::ForEachN(d_numbers, size, SquareAndAdd(10));

    // Check for errors during kernel execution
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }

    // Copy result from device to host
    hipMemcpy(results.data(), d_numbers, size * sizeof(int), hipMemcpyDeviceToHost);

    // Check results
    std::cout << "\n INPUT  OUTPUT\n";
    std::cout << "--------------\n";
    for (int i = 0; i < 20; ++i) {
        std::cout << std::setw(6) << numbers[i]
                  << std::setw(6) << results[i] << "\n";
    }
    
    // Free device memory and return
    hipFree(d_numbers);
    return 0;
}
