#include <iomanip>
#include <iostream>
#include <numeric>
#include <vector>
#include <hipcub/hipcub.hpp>

struct Square {
    __device__ void operator() (int& x) { x *= x; };
};

using blocker = hipcub::WarpLoad<int, 4, hipcub::WARP_LOAD_DIRECT, 32>;

int main() {
    // Useful values
    constexpr int size = 10000;

    // Initialize host vector
    std::vector<int> numbers(size);
    std::vector<int> squares(size);
    std::iota(numbers.begin(), numbers.end(), 0);
    
    // Allocate device memory and copy from host
    void* p_numbers;
    hipMalloc(&p_numbers, size * sizeof(int));
    int* d_numbers = static_cast<int*>(p_numbers);
    hipMemcpy(d_numbers, numbers.data(), size * sizeof(int), hipMemcpyHostToDevice);

    // DeviceFor application
    hipcub::DeviceFor::ForEachN(d_numbers, size, Square());

    // Check that execution went well, or print error string
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }

    // Copy result from device to host
    hipMemcpy(squares.data(), d_numbers, size * sizeof(int), hipMemcpyDeviceToHost);

    // Check results
    std::cout << "\n INPUT  OUTPUT\n";
    std::cout << "--------------\n";
    for (int i = 0; i < 20; ++i) {
        std::cout << std::setw(6) << numbers[i]
                  << std::setw(6) << squares[i] << "\n";
    }
    
    // Free device memory and return
    hipFree(d_numbers);
    return 0;
}
