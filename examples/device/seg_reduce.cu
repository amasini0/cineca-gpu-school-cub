#include <iomanip>
#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

constexpr int num_segments = 4;

int main() {
    // Useful values
    constexpr int num_items = 100;

    // Allocate host vector
    std::vector<int> items(num_items);
    std::vector<int> offsets(num_segments + 1);
    std::vector<int> sums(num_segments);
    std::vector<int> mins(num_segments);

    // Fill input vector
    std::iota(items.begin(), items.end(), 1);

    // Fill offsets
    int segment_length = num_items / num_segments;
    for (int i = 0; i < num_segments; ++i) {
        offsets[i] = i * segment_length;
    }
    offsets[num_segments] = num_items;

    // Allocate arrays on device
    void *p_items, *p_offsets, *p_sums, *p_mins;
    hipMalloc(&p_items, num_items*sizeof(int));
    hipMalloc(&p_offsets, (num_segments + 1) * sizeof(int));
    hipMalloc(&p_sums, num_segments*sizeof(int));
    hipMalloc(&p_mins, num_segments*sizeof(int));
    int *d_items = static_cast<int*>(p_items);
    int *d_offsets = static_cast<int*>(p_offsets);
    int *d_sums  = static_cast<int*>(p_sums);
    int *d_mins  = static_cast<int*>(p_mins);

    // Copy items to device
    hipMemcpy(d_items, items.data(), num_items*sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), (num_segments + 1) * sizeof(int), hipMemcpyHostToDevice);

    // SUM --------------------------------------------------- //
    // Determine temporary device storage requirements
    void* p_temp_storage_sum = nullptr;
    size_t temp_storage_sum_bytes = 0;
    hipcub::DeviceSegmentedReduce::Sum(
        p_temp_storage_sum, temp_storage_sum_bytes, 
        d_items, d_sums, num_segments, d_offsets, d_offsets + 1);

    // Allocate required temporary storage
    hipMalloc(&p_temp_storage_sum, temp_storage_sum_bytes);

    // Perform the reduction
    hipcub::DeviceSegmentedReduce::Sum(
        p_temp_storage_sum, temp_storage_sum_bytes, 
        d_items, d_sums, num_segments, d_offsets, d_offsets + 1);

    // MIN --------------------------------------------------- //
    // Determine temporary device storage requirements
    void* p_temp_storage_min = nullptr;
    size_t temp_storage_min_bytes = 0;
    hipcub::DeviceSegmentedReduce::Min(
        p_temp_storage_min, temp_storage_min_bytes, 
        d_items, d_mins, num_segments, d_offsets, d_offsets + 1);

    // Allocate required temporary storage
    hipMalloc(&p_temp_storage_min, temp_storage_min_bytes);

    // Perform the reduction
    hipcub::DeviceSegmentedReduce::Min(
        p_temp_storage_min, temp_storage_min_bytes, 
        d_items, d_mins, num_segments, d_offsets, d_offsets + 1);

    // Check results
    hipMemcpy(sums.data(), d_sums, num_segments * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(mins.data(), d_mins, num_segments * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Segment aggregates\n";
    std::cout << "   SUM    MIN\n------------\n";
    for (int s = 0; s < num_segments; ++s) {
        std::cout << std::setw(6) << sums[s] << " "
                  << std::setw(6) << mins[s] << "\n";
    }

    hipFree(d_items);
    hipFree(d_offsets);
    hipFree(d_sums);
    hipFree(d_mins);
    hipFree(p_temp_storage_sum);
    hipFree(p_temp_storage_min);
    return 0;
}
