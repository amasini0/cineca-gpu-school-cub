#include <iomanip>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>
#include <hipcub/hipcub.hpp>
#include <cub/device/device_select.cuh>

// Predicate for if
struct LessThan {
    int ref_value;
    explicit LessThan(int x) : ref_value{x} {}
    __device__ bool operator() (const int& x) { return x < ref_value; }
};

int main() {
    // Useful values
    constexpr int num_items = 60;
    constexpr int threshold = 5;

    // Allocate host vector
    std::vector<int> items(num_items);
    std::vector<int> uniqs(num_items);
    
    // Fill vector with random sequence of values
    std::random_device rd;
    std::uniform_int_distribution<int> random_value(0,9);
    for (auto& el: items) {
        el = random_value(rd);
    }

    // Allocate memory on device
    void *p_items, *p_uniqs;
    hipMalloc(&p_items, num_items * sizeof(int));
    hipMalloc(&p_uniqs, num_items * sizeof(int));
    int* d_items = static_cast<int*>(p_items);
    int* d_uniqs = static_cast<int*>(p_uniqs);

    // Copy items to device
    hipMemcpy(d_items, items.data(), num_items * sizeof(int), hipMemcpyHostToDevice);
    
    // Allocate pointer to store number of not selected elements
    void *p_num_selected;
    hipMalloc(&p_num_selected, sizeof(int));
    int* d_num_selected = static_cast<int*>(p_num_selected);

    // Determine temporary storage requirements
    void *p_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::If(
        p_temp_storage, temp_storage_bytes,
        d_items, d_uniqs, d_num_selected, num_items, LessThan(threshold));

    // Allocate temporary storage
    hipMalloc(&p_temp_storage, temp_storage_bytes);

    // Run selection
    hipcub::DeviceSelect::If(
        p_temp_storage, temp_storage_bytes,
        d_items, d_uniqs, d_num_selected, num_items, LessThan(threshold));

    // Copy number of uniques back to host
    int num_selected = 0;
    hipMemcpy(&num_selected, d_num_selected, sizeof(int), hipMemcpyDeviceToHost);

    // Copy back output array of uniques
    hipMemcpy(uniqs.data(), d_uniqs, num_items*sizeof(int), hipMemcpyDeviceToHost);

    // Check results
    std::cout << "Number of elements less than " << threshold 
              << " is: " << num_selected << "\n";
    std::cout << "Selected elements: [ ";
    for (int s = 0; s < num_selected; ++s) {
        std::cout << uniqs[s] << " ";
    }
    std::cout << "]\n";
    
    // Release resources and finish execution
    hipFree(d_items);
    hipFree(d_uniqs);
    hipFree(d_num_selected);
    hipFree(p_temp_storage);
    return 0;
}
