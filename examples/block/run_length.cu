#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <numeric>
#include <random>
#include <vector>
#include <hipcub/hipcub.hpp>

constexpr int num_blocks = 2;
constexpr int block_dim = 32;
constexpr int items_per_thread = 4;
constexpr int runs_per_thread = items_per_thread; // must be same as items_per_thread

/*--------------ONLY MODIFY THIS FUNCTION --------------------------------------*/        
using DecodeT = hipcub::BlockRunLengthDecode<int, block_dim, runs_per_thread, items_per_thread>;
using StoreT = hipcub::BlockStore<int, block_dim, items_per_thread, hipcub::BLOCK_STORE_VECTORIZE>;

__global__ void blockDecode(int* sizes, int* values, int* lengths, int* output) {
    __shared__ DecodeT::TempStorage dc_temp;
    __shared__ StoreT::TempStorage  st_temp;

    int thread_values[runs_per_thread] = { 0 }; // init to zero
    int thread_lengths[runs_per_thread] = { 0 }; // init to zero

    // Load data from values and lengths inside thread-local arrays
    // Since we (probably) have more threads than required, due to the fact that
    // a thread handles more than one run, we will give the extra threads zero-filled
    // arrays (both for values and lengths, but the most important is lengths), so
    // that they will not influence the decoding
    int block_run_offset = (blockIdx.x != 0) ? sizes[blockIdx.x-1] : 0;
    int block_runs = sizes[blockIdx.x];
    int thread_run_offset = threadIdx.x * runs_per_thread;
    
    for (int i = 0; i < runs_per_thread; ++i) {
        int block_run_idx = thread_run_offset + i;
        int global_run_idx = block_run_offset + block_run_idx;
        if (block_run_idx < block_runs) {
            thread_values[i] = values[global_run_idx];
            thread_lengths[i] = lengths[global_run_idx];  
        }
    }

    // Initialize decoder and get total decoded size from it (not used further)
    // As a check, by construction total_decoded_size should be equal to items_per_thread * block_dim
    int total_decoded_size = 0;
    DecodeT decoder(dc_temp, thread_values, thread_lengths, total_decoded_size);
    
    // Run decoding of a batch of elements (the number is the width of the window, which 
    // depends on the template parameters). 
    // Generally, this should be in a while loop, since more than one batch of decoded 
    // elements may be required to decode all the sequence. 
    // In this case, we specialized the template to get the window size equal to the number
    // of elements to decode (per block), i.e. items_per_thread * 32 (threads in a block)
    // thus we only need one decoding pass to get them all. 
    int decoded_items[items_per_thread];
    decoder.RunLengthDecode(decoded_items, /* offset */ 0);

    // Store results in the correct output position
    const int block_output_offset = blockIdx.x * blockDim.x * items_per_thread;
    StoreT(st_temp).Store(output + block_output_offset, decoded_items);
}


/*--------------DO NOT CHANGE THIS PART OF THE CODE ----------------------------*/
using DiscT = hipcub::BlockDiscontinuity<int, block_dim>;
using ScanT = hipcub::BlockScan<int, block_dim, hipcub::BLOCK_SCAN_WARP_SCANS>;
using LoadT = hipcub::BlockLoad<int, block_dim, items_per_thread, hipcub::BLOCK_LOAD_VECTORIZE>;

__global__ void blockEncode (int* input, int* sizes, int* values, int* lengths) {
    // Allocate shared memory
    __shared__ LoadT::TempStorage ld_temp;
    __shared__ DiscT::TempStorage ds_temp;
    __shared__ ScanT::TempStorage sc_temp;

    // Declare thread-local data
    const int block_offset = blockIdx.x * blockDim.x * items_per_thread;
    int thread_data[items_per_thread];
    int thread_discont_mask[items_per_thread];
    int thread_scanned_mask[items_per_thread];
    
    // Load thread-local data
    LoadT(ld_temp).Load(input + block_offset, thread_data);
    
    // Compute discontinuity mask (puts a 1 at first element of each run)
    DiscT(ds_temp).FlagHeads(thread_discont_mask, thread_data, hipcub::Inequality());

    // Compute inclusive prefix sum of the ones inside the mask
    ScanT(sc_temp).InclusiveSum(thread_discont_mask, thread_scanned_mask);

    // If lengths or values are nullptr, only output size required on each block
    if (values == nullptr || lengths == nullptr) {
        if (threadIdx.x == (blockDim.x -1)) {
            sizes[blockIdx.x] = thread_scanned_mask[items_per_thread - 1]; 
        }
        return;
    }

    // The following executes only if values & lenghts point to valid memory
    // and sizes array is already filled (by a previous call to block_rle)

    // Get offset for writing in output array (equals the number of runs 
    // assigned to the prev block if block_idx != 0, else 0)
    int offset = (blockIdx.x != 0) ? sizes[blockIdx.x-1] : 0;

    // For each element of the thread, if it is the start of a run (i.e. if it has
    // a 1 in the discontinuity mask) write its value in correct position (i.e.
    // the one written in the scanned mask + offset from prev block - 1), and 
    // store the item's idx (thread_idx * items_per_thread + item_in_thread_idx)
    // in the lengths array to later get run length.
    for (int i = 0; i < items_per_thread; ++i) {
        if (thread_discont_mask[i] == 1) {
            int item_idx = threadIdx.x * items_per_thread + i;
            int out_idx = offset + thread_scanned_mask[i] - 1;
            values[out_idx]  = thread_data[i];
            lengths[out_idx] = item_idx;
        }
    }

    // Compute effective lenghts as difference of adjacent starting item indices 
    // obtained in the previous block (stored in lengths).
    // Only the first size-1 (size is the number of runs associated to the block)
    // are working, the others are idle.
    // Each active thread (except the last one) computes difference of starting 
    // indices of its run and the following run, to obtain the run's length.
    // Last active thread (idx = size-1) computes lenght as difference between 
    // its starting index and the index of the last block's item.
    int size = sizes[blockIdx.x];
    for (int thread_lid = threadIdx.x; thread_lid < size; thread_lid += blockDim.x) {
        int thread_gid = offset + thread_lid;
        int run_length = (thread_lid < (size - 1)) ? lengths[thread_gid + 1] - lengths[thread_gid]
                                                   : blockDim.x * items_per_thread - lengths[thread_gid];
        __syncthreads();
        lengths[thread_gid] = run_length;
    }
}

int main() {
    // Useful values
    constexpr int full_size = num_blocks * block_dim * items_per_thread;
    
    // Allocate input, output and sizes vectors on host
    std::vector<int> input(full_size);
    std::vector<int> output(full_size);
    std::vector<int> sizes(num_blocks);

    // Populate input with random sequences of integers
    std::random_device rd;
    std::uniform_int_distribution<int> rand_length(1,9);
    std::uniform_int_distribution<int> rand_value(0,9);

    int i = 0, l = 0, v = 0;
    while (i < full_size) {
        l = std::min(rand_length(rd), full_size - i); // Avoids going out of bounds
        v = rand_value(rd);
        for (int j = 0; j < l; ++j) input[i+j] = v;
        i += l;
    }

    // Allocate input, output and sizes arrays on device
    void *p_input, *p_output, *p_sizes;
    hipMalloc(&p_input, full_size * sizeof(int));
    hipMalloc(&p_output, full_size * sizeof(int));
    hipMalloc(&p_sizes, num_blocks * sizeof(int));
    int* d_input  = static_cast<int*>(p_input);
    int* d_output = static_cast<int*>(p_output);
    int* d_sizes  = static_cast<int*>(p_sizes);
    
    // Copy input to device
    hipMemcpy(d_input, input.data(), full_size * sizeof(int), hipMemcpyHostToDevice);
    
    // Get sizes from blockEncode(sizes of encoded seqs on each block)
    blockEncode<<<num_blocks, block_dim>>>(d_input, d_sizes, nullptr, nullptr);
    
    // Check for errors during kernel execution
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << "\n";
    }

    // Copy sizes array to host
    hipMemcpy(sizes.data(), d_sizes, num_blocks * sizeof(int), hipMemcpyDeviceToHost);
   
    // Compute total size of output
    const int encoded_size = std::accumulate(sizes.begin(), sizes.end(), 0);

    // Allocate values and lengths vectors on host
    std::vector<int> values(encoded_size);
    std::vector<int> lengths(encoded_size);

    // Allocate values and lengths arrays on device
    void *p_values, *p_lengths;
    hipMalloc(&p_values, encoded_size * sizeof(int));
    hipMalloc(&p_lengths, encoded_size * sizeof(int));
    int* d_values = static_cast<int*>(p_values);
    int* d_lengths = static_cast<int*>(p_lengths);

    // Run encoding (this time for real)
    blockEncode<<<num_blocks, block_dim>>>(d_input, d_sizes, d_values, d_lengths);
    
    // Check for errors during kernel execution
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << "\n";
    }

    // Copy encoded sequences to host for printing
    hipMemcpy(values.data(), d_values, encoded_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(lengths.data(), d_lengths, encoded_size * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Encoded sequence length: " << std::setw(3) << encoded_size << "\n";
    std::cout << "\n VAL  RUN\n";
    int offset = 0;
    for (int b = 0; b < num_blocks; ++b) {
        const int size = sizes[b];
        std::cout << "--------- Block: " << std::setw(2) << b << "\n";
        for (int i = 0; i < size; ++i) {
            std::cout << std::setw(4) << values[offset + i] << " "
                      << std::setw(4) << lengths[offset + i] << "\n";
        }
        offset += size;
    }

    // Run decoding (use single block, it's easier)
    blockDecode<<<num_blocks, block_dim>>>(d_sizes, d_values, d_lengths, d_output);

    // Check for errors during kernel execution
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << "\n";
    }
    
    // Copy output (decoded) sequence to host for checking
    hipMemcpy(output.data(), d_output, full_size * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "\n INPUT OUTPUT\n-------------\n";
    for (int i = 0; i < 20; ++i) {
        std::cout << std::setw(6) << input[i] << " "
                  << std::setw(6) << output[i] << "\n";
    }

    // Check that input and output match
    std::cout << "\n\n";
    bool mismatch = false;
    for (int i = 0; i < full_size; ++i) {
        if (input[i] != output[i]) {
            mismatch = true;
            std::cout << "Mismatch at element: " << std::setw(4) << i << "\n";
            std::cout << "Decoding FAILED\n";
            break;
        }
    }
    if (!mismatch) std::cout << "No mismatches found\nDecoding SUCCEDED\n";

    // Release resources and finish execution
    hipFree(d_input);
    hipFree(d_sizes);
    hipFree(d_output);
    hipFree(d_values);
    hipFree(d_lengths);
    return 0;
}
